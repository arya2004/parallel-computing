#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void monteCarloPiKernel(unsigned int *count, unsigned int N, unsigned int seed) {
    
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = gridDim.x * blockDim.x;

    unsigned int localCount = 0;

    // random generator
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    for (unsigned int i = tid; i < N; i += stride){


        float x = hiprand_uniform(&state);
        float y = hiprand_uniform(&state);
        if (x * x + y * y <= 1.0f){
            localCount++;
        }
           
    }

    atomicAdd(count, localCount);
}

int main() {
    unsigned int N = 1 << 26; // 4m points


    unsigned int *d_count, h_count = 0;
    hipMalloc(&d_count, sizeof(unsigned int));

    hipMemcpy(d_count, &h_count, sizeof(unsigned int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = 128;

    monteCarloPiKernel<<<blocks, threads>>>(d_count, N, time(NULL));


    hipMemcpy(&h_count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    float pi_estimate = 4.0f * h_count / N;



    cout << "Estimated Pi = " << pi_estimate << endl;


    hipFree(d_count);
    return 0;
}
