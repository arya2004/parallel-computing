#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 25

__global__ void vectorAdd(float *a, float *b, float *c) {
    int idx = threadIdx.x;
    if (idx < N) c[idx] = a[idx] + b[idx];
}

int main() {
    float h_a[N] = { 1,  2,  3,  4,  5,  6,  7,  8,  9, 10,
                     11, 12, 13, 14, 15, 16, 17, 18, 19, 20,
                     21, 22, 23, 24, 25 };
    float h_b[N] = { 25, 24, 23, 22, 21, 20, 19, 18, 17, 16,
                     15, 14, 13, 12, 11, 10,  9,  8,  7,  6,
                      5,  4,  3,  2,  1 };
    float h_c[N];
    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    vectorAdd<<<1, N>>>(d_a, d_b, d_c);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float gpuMs = 0.0f;
    hipEventElapsedTime(&gpuMs, start, stop);
    printf("GPU kernel time:   %.3f ms\n", gpuMs);

    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    clock_t cpuStart = clock();
    for (int i = 0; i < N; i++) {
        h_c[i] = h_a[i] + h_b[i];
    }
    clock_t cpuEnd = clock();
    double cpuMs = (double)(cpuEnd - cpuStart) / CLOCKS_PER_SEC * 1000.0;
    printf("CPU serial time:   %.3f ms\n", cpuMs);

    for (int i = 0; i < N; i++) {
        printf("%.0f + %.0f = %.0f\n", h_a[i], h_b[i], h_c[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
